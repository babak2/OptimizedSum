#include "hip/hip_runtime.h"
﻿
/*
Program: OptimizedSum - Demonstrating CPU vs. CUDA Summation
Author: Babak Mahdavi Ardestani
Date: October 26, 2022
*/


#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cstdlib>
#include <ctime>

// CPU version to calculate the sum
int cpuSum(int* array, int size) {
    int sum = 0;
    for (int i = 0; i < size; i++) {
        sum += array[i];
    }
    return sum;
}

#ifdef __HIPCC__
// CUDA version to calculate the sum
__global__ void cudaSum(int* array, int* result, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int localSum = 0;
    while (tid < size) {
        localSum += array[tid];
        tid += blockDim.x * gridDim.x;
    }
    atomicAdd(result, localSum);
}
#endif

int main() {
    const int size = 10000000;  // Size of the array
    int* h_array = new int[size];
    int* d_array, * d_result;

    // Initialize the array with random values
    srand(static_cast<unsigned>(time(0)));
    for (int i = 0; i < size; i++) {
        h_array[i] = rand() % 100;
    }

 

    // CPU version
    clock_t start_time_cpu = clock();
    int cpu_result = cpuSum(h_array, size);
    clock_t end_time_cpu = clock();
    double cpu_time = double(end_time_cpu - start_time_cpu) / CLOCKS_PER_SEC;
    std::cout << "CPU Sum: " << cpu_result << " - Time: " << cpu_time << " seconds\n";

    // CUDA version
    hipMalloc((void**)&d_array, size * sizeof(int));
    hipMalloc((void**)&d_result, sizeof(int));
    hipMemcpy(d_array, h_array, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    clock_t start_time = clock();
    int numThreads = 256;
    int numBlocks = (size + numThreads - 1) / numThreads;
    cudaSum << <numBlocks, numThreads >> > (d_array, d_result, size);

    hipDeviceSynchronize();
    int cuda_result;
    hipMemcpy(&cuda_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    clock_t end_time = clock();
    double cuda_time = double(end_time - start_time) / CLOCKS_PER_SEC;
    std::cout << "CUDA Sum: " << cuda_result << " - Time: " << cuda_time << " seconds\n";

    hipFree(d_array);
    hipFree(d_result);
    delete[] h_array;

    std::cout << "CPU Time: " << cpu_time << " seconds\n";
    std::cout << "CUDA Time: " << cuda_time << " seconds\n";
    std::cout << "Speedup: " << cpu_time / cuda_time << "x\n";

    return 0;
}
